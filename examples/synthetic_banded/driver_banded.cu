#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <cmath>

#include <cusp/io/matrix_market.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>

#include <spike/solver.h>
#include <spike/spmv.h>


// -----------------------------------------------------------------------------
// Macro to obtain a random number between two specified values
// -----------------------------------------------------------------------------
#define RAND(L,H)  ((L) + ((H)-(L)) * (float)rand()/(float)RAND_MAX)


// -----------------------------------------------------------------------------
// Typedefs
// -----------------------------------------------------------------------------
typedef double REAL;
typedef float  PREC_REAL;

typedef typename cusp::csr_matrix<int, REAL, cusp::device_memory> Matrix;
typedef typename cusp::array1d<REAL, cusp::device_memory>         Vector;

typedef typename spike::Solver<Vector, PREC_REAL>                 SpikeSolver;
typedef typename spike::SpmvCusp<Matrix>                          SpmvFunctor;

typedef typename cusp::coo_matrix<int, REAL, cusp::host_memory>   MatrixCooH;
typedef typename cusp::array1d<REAL, cusp::host_memory>           VectorH;


// -----------------------------------------------------------------------------
using std::cout;
using std::cin;
using std::endl;
using std::string;
using std::vector;


// -----------------------------------------------------------------------------
// Definitions for SimpleOpt and SimpleGlob
// -----------------------------------------------------------------------------
#include <SimpleOpt/SimpleOpt.h>

// ID values to identify command line arguments
enum {OPT_HELP, OPT_VERBOSE, OPT_PART,
      OPT_RTOL, OPT_ATOL, OPT_MAXIT,
      OPT_DROPOFF_FRAC, 
      OPT_BAND,
      OPT_OUTFILE, OPT_PRECOND,
      OPT_KRYLOV, OPT_SAFE_FACT};

// Table of CSimpleOpt::Soption structures. Each entry specifies:
// - the ID for the option (returned from OptionId() during processing)
// - the option as it should appear on the command line
// - type of the option
// The last entry must be SO_END_OF_OPTIONS
CSimpleOptA::SOption g_options[] = {
	{ OPT_PART,          "-p",                   SO_REQ_CMB },
	{ OPT_PART,          "--num-partitions",     SO_REQ_CMB },
	{ OPT_RTOL,          "-t",                   SO_REQ_CMB },
	{ OPT_RTOL,          "--tolerance",          SO_REQ_CMB },
	{ OPT_RTOL,          "--relTol",             SO_REQ_CMB },
	{ OPT_ATOL,          "--absTol",             SO_REQ_CMB },
	{ OPT_MAXIT,         "-i",                   SO_REQ_CMB },
	{ OPT_MAXIT,         "--max-num-iterations", SO_REQ_CMB },
	{ OPT_DROPOFF_FRAC,  "-d",                   SO_REQ_CMB },
	{ OPT_DROPOFF_FRAC,  "--drop-off-fraction",  SO_REQ_CMB },
	{ OPT_BAND,          "-b",                   SO_MULTI   },
	{ OPT_BAND,          "--banded-synthetic",   SO_MULTI   },
	{ OPT_OUTFILE,       "-o",                   SO_REQ_CMB },
	{ OPT_OUTFILE,       "--output-file",        SO_REQ_CMB },
	{ OPT_PRECOND,       "--precond-method",     SO_REQ_CMB },
	{ OPT_KRYLOV,        "-k",                   SO_REQ_CMB },
	{ OPT_KRYLOV,        "--krylov-method",      SO_REQ_CMB },
	{ OPT_SAFE_FACT,     "--safe-fact",          SO_NONE    },
	{ OPT_VERBOSE,       "-v",                   SO_NONE    },
	{ OPT_VERBOSE,       "--verbose",            SO_NONE    },
	{ OPT_HELP,          "-?",                   SO_NONE    },
	{ OPT_HELP,          "-h",                   SO_NONE    },
	{ OPT_HELP,          "--help",               SO_NONE    },
	SO_END_OF_OPTIONS
};

// Color to print
enum TestColor {COLOR_NO = 0,
                COLOR_RED,
                COLOR_GREEN} ;

// -----------------------------------------------------------------------------
// Forward declarations.
// -----------------------------------------------------------------------------
void ShowUsage();
void spikeSetDevice();
bool GetProblemSpecs(int             argc, 
                     char**          argv,
                     int&            N,
                     int&            k,
                     REAL&           d,
                     string&         fileSol,
                     int&            numPart,
                     bool&           verbose,
                     spike::Options& opts);

void GetBandedMatrix(int N, int k, REAL d, Matrix& A);
void GetRhsVector(const Matrix& A, Vector& b, Vector& x_target);
void PrintStats(bool               success,
                const SpikeSolver& mySolver,
                const SpmvFunctor& mySpmv);

class OutputItem
{
public:
	OutputItem(std::ostream &o): m_o(o), m_additional_item_count(4) {}

	int           m_additional_item_count;

	template <typename T>
	void operator() (T item, TestColor c = COLOR_NO) {
		m_o << "<td style=\"border-style: inset;\">\n";
		switch (c)
		{
			case COLOR_RED:
				m_o << "<p> <FONT COLOR=\"Red\">" << item << " </FONT> </p>\n";
				break;

			case COLOR_GREEN:
				m_o << "<p> <FONT COLOR=\"Green\">" << item << " </FONT> </p>\n";
				break;

			default:
				m_o << "<p> " << item << " </p>\n";
				break;
		}
		m_o << "</td>\n";
	}
private:
	std::ostream &m_o;
};


// -----------------------------------------------------------------------------
// MAIN
// -----------------------------------------------------------------------------
int main(int argc, char** argv) 
{
	// Set up the problem to be solved.
	int            pN;
	int            pk;
	REAL           pd;
	string         fileSol;
	int            numPart;
	bool           verbose;
	spike::Options opts;

	opts.trackReordering = false;
	opts.variableBandwidth = false;
	opts.factMethod = spike::LU_UL;
	opts.performReorder = false;
	opts.applyScaling = false;

	if (!GetProblemSpecs(argc, argv, pN, pk, pd, fileSol, numPart, verbose, opts))
		return 1;

	// Get the device with most available memory.
	spikeSetDevice();

	// Get matrix and rhs.
	Matrix A;
	Vector b;
	Vector x_target;
	Vector delta_x_target;

	GetBandedMatrix(pN, pk, pd, A);
	GetRhsVector(A, b, x_target);

	// Create the SPIKE Solver object and the SPMV functor. Perform the solver
	// setup, then solve the linear system using a 0 initial guess.
	// Set the initial guess to the zero vector.
	SpikeSolver  mySolver(numPart, opts);
	SpmvFunctor  mySpmv(A);
	Vector x(A.num_rows, 0);

	bool success = false;

	OutputItem outputItem(cout);
	cout << "<tr valign=top>" << endl;

	outputItem(pN);
	outputItem(pk);
	outputItem(pd);
	outputItem(numPart);

	try {
		mySolver.setup(A);
		success = mySolver.solve(mySpmv, b, x);
	} catch (const std::bad_alloc&) {
		outputItem ("OoM (in setup stage)", COLOR_RED);

		for (int i = 0; i < outputItem.m_additional_item_count; i++)
			outputItem("");

		cout << "</tr>" << endl;

		return 1;
	}

	spike::Stats stats = mySolver.getStats();

	// Reason why cannot solve (for unsuccessful solving only)
	if (success)
		outputItem ( "OK");
	else
		outputItem ( "NConv", COLOR_RED);

	// Total time for setup
	outputItem( stats.timeSetup);
	// Number of iterations to converge
	outputItem( stats.numIterations);
	// Total time for Krylov solve
	outputItem( stats.timeSolve);
	// Total amount of time
	outputItem( stats.timeSetup + stats.timeSolve);

	cout << "</tr>" << endl;

	// Write solution file and print solver statistics.
	if (fileSol.length() > 0)
		cusp::io::write_matrix_market_file(x, fileSol);

	// Calculate the actual residual and its norm.
	/*
	if (verbose) {
		PrintStats(success, mySolver, mySpmv);

		Vector r(A.num_rows);
		mySpmv(x, r);
		cusp::blas::axpby(b, r, r, REAL(1.0), REAL(-1.0));
		cout << "|b - A*x|      = " << cusp::blas::nrm2(r) << endl;
		cout << "|b|            = " << cusp::blas::nrm2(b) << endl;	
		cout << "|x_target|     = " << cusp::blas::nrm2(x_target) << endl;
		delta_x_target.resize(A.num_rows);
		cusp::blas::axpby(x_target, x, delta_x_target, REAL(1.0), REAL(-1.0));
		cout << "|x_target - x| = " << cusp::blas::nrm2(delta_x_target) << endl;
	} else {
		spike::Stats stats = mySolver.getStats();
		printf("%d,%d,%d,%g,%g\n", success, pN, pk, pd, stats.timeSetup + stats.timeSolve);
	}*/

	return 0;
}


// -----------------------------------------------------------------------------
// spikeSetDevice()
//
// This function sets the active device to be the one with maximum available
// space.
// -----------------------------------------------------------------------------
void spikeSetDevice() {
	int deviceCount = 0;
	
	if (hipGetDeviceCount(&deviceCount) != hipSuccess || deviceCount <= 0) {
		std::cerr << "There is no available device." << endl;
		exit(-1);
	}

	size_t max_free_size = 0;
	int max_idx = 0;
	for (int i=0; i < deviceCount; i++) {
		hipSetDevice(i);
		size_t free_size = 0, total_size = 0;
		if (hipMemGetInfo(&free_size, &total_size) == hipSuccess)
			if (max_free_size < free_size) {
				max_idx = i;
				max_free_size = free_size;
			}
	}

	// std::cerr << "Use device: " << max_idx << endl;
	// hipSetDevice(max_idx);
	hipSetDevice(max_idx);
}

// -------------------------------------------------------------------
// GetRhsVector()
//
// This function generates a RHS vector of appropriate dimension. We
// use the method of manufactured solution, meaning we set
//    b = A * x
// for a known "solution" vector x.
// -------------------------------------------------------------------
void
GetRhsVector(const Matrix& A, Vector& b, Vector& x_target)
{
	int     N = A.num_rows;
	b.resize(N, (REAL)1.0);
	// Create a desired solution vector (on the host), then copy it
	// to the device.
	/*
	int     N = A.num_rows;
	REAL    dt = 1.0/(N-1);
	REAL    max_val = 100.0;

	VectorH xh(N);

	for (int i = 0; i < N; i++) {
		REAL t = i *dt;
		xh[i] = 4 * max_val * t * (1 - t);
	}

	x_target = xh;
	
	// Calculate the RHS vector.
	b.resize(N);
	cusp::multiply(A, x_target, b);  */
	////cusp::io::write_matrix_market_file(b, "b.mtx");
}

// -----------------------------------------------------------------------------
// GetProblemSpecs()
//
// This function parses the specified program arguments and sets up the problem
// to be solved.
// -----------------------------------------------------------------------------
bool
GetProblemSpecs(int             argc, 
                char**          argv,
                int&            N,
                int&            k,
                REAL&           d,
                string&         fileSol,
                int&            numPart,
                bool&           verbose,
                spike::Options& opts)
{
	numPart = -1;
	verbose = false;

	// Create the option parser and pass it the program arguments and the array
	// of valid options. Then loop for as long as there are arguments to be
	// processed.
	CSimpleOptA args(argc, argv, g_options);

	while (args.Next()) {
		// Exit immediately if we encounter an invalid argument.
		if (args.LastError() != SO_SUCCESS) {
			cout << "Invalid argument: " << args.OptionText() << endl;
			ShowUsage();
			return false;
		}

		// Process the current argument.
		switch (args.OptionId()) {
			case OPT_HELP:
				ShowUsage();
				return false;
			case OPT_PART:
				numPart = atoi(args.OptionArg());
				break;
			case OPT_RTOL:
				opts.relTol = atof(args.OptionArg());
				break;
			case OPT_ATOL:
				opts.absTol = atof(args.OptionArg());
				break;
			case OPT_MAXIT:
				opts.maxNumIterations = atoi(args.OptionArg());
				break;
			case OPT_DROPOFF_FRAC:
				opts.dropOffFraction = atof(args.OptionArg());
				break;
			case OPT_BAND:
				{
					char **mArgs = args.MultiArg(3);
					if (!mArgs) {
						return false;
					}
					N = atoi(mArgs[0]);
					k = atoi(mArgs[1]);
					d = atof(mArgs[2]);

					break;
				}
			case OPT_OUTFILE:
				fileSol = args.OptionArg();
				break;
			case OPT_VERBOSE:
				verbose = true;
				break;
			case OPT_PRECOND:
				{
					string precond = args.OptionArg();
					std::transform(precond.begin(), precond.end(), precond.begin(), ::toupper);
					if (precond == "0" || precond == "SPIKE")
						opts.precondType = spike::Spike;
					else if(precond == "1" || precond == "BLOCK")
						opts.precondType = spike::Block;
					else if(precond == "2" || precond == "NONE")
						opts.precondType = spike::None;
					else
						return false;
				}
				break;
			case OPT_KRYLOV:
				{
					string kry = args.OptionArg();
					std::transform(kry.begin(), kry.end(), kry.begin(), ::toupper);
					if (kry == "0" || kry == "BICGSTAB_C")
						opts.solverType = spike::BiCGStab_C;
					else if (kry == "1" || kry == "GMRES_C")
						opts.solverType = spike::GMRES_C;
					else if (kry == "2" || kry == "CG_C")
						opts.solverType = spike::CG_C;
					else if (kry == "3" || kry == "CR_C")
						opts.solverType = spike::CR_C;
					else if (kry == "4" || kry == "BICGSTAB1")
						opts.solverType = spike::BiCGStab1;
					else if (kry == "5" || kry == "BICGSTAB2")
						opts.solverType = spike::BiCGStab2;
					else if (kry == "6" || kry == "BICGSTAB")
						opts.solverType = spike::BiCGStab;
					else if (kry == "7" || kry == "MINRES")
						opts.solverType = spike::MINRES;
					else
						return false;
				}
				break;
			case OPT_SAFE_FACT:
				opts.safeFactorization = true;
				break;
		}
	}

	// If the number of partitions was not defined, show usage and exit.
	if (numPart <= 0) {
		cout << "The number of partitions must be specified." << endl << endl;
		ShowUsage();
		return false;
	}

	// If no reordering, force using constant bandwidth.
	if (!opts.performReorder)
		opts.variableBandwidth = false;

	// If using variable bandwidth, force using LU factorization.
	if (opts.variableBandwidth)
		opts.factMethod = spike::LU_only;

	// Print out the problem specifications.
	if (verbose) {
		cout << endl;
		cout << "Problem size: " << N << " "<<k <<" "<<d<<endl;
		if (fileSol.length() > 0)
			cout << "Sol file:    " << fileSol << endl;
		cout << "Iterative solver: ";
		switch (opts.solverType) {
		case spike::BiCGStab_C:
			cout << "BiCGStab (Cusp)" << endl; break;
		case spike::GMRES_C:
			cout << "GMRES (Cusp)" << endl; break;
		case spike::CG_C:
			cout << "CG (Cusp)" << endl; break;
		case spike::CR_C:
			cout << "CR (Cusp)" << endl; break;
		case spike::BiCGStab1:
			cout << "BiCGStab1 (Spike::GPU)" << endl; break;
		case spike::BiCGStab2:
			cout << "BiCGStab2 (Spike::GPU)" << endl; break;
		case spike::BiCGStab:
			cout << "BiCGStab (Spike::GPU)" << endl; break;
		case spike::MINRES:
			cout << "MINRES (Spike::GPU)" << endl; break;
		}
		cout << "Relative tolerance: " << opts.relTol << endl;
		cout << "Absolute tolerance: " << opts.absTol << endl;
		cout << "Max. iterations: " << opts.maxNumIterations << endl;
		cout << "Preconditioner: ";
		switch (opts.precondType) {
		case spike::Spike:
			cout << "SPIKE" << endl; break;
		case spike::Block:
			cout << "BLOCK DIAGONAL" << endl; break;
		case spike::None:
			cout << "NONE" << endl; break;
		}
		if (opts.precondType != spike::None) {
			cout << "Using " << numPart << (numPart ==1 ? " partition." : " partitions.") << endl;
			cout << "Factorization method: LU - UL" << endl;
			if (opts.dropOffFraction > 0)
				cout << "Drop-off fraction: " << opts.dropOffFraction << endl;
			else
				cout << "No drop-off." << endl;
			cout << (opts.safeFactorization ? "Use safe factorization." : "Use non-safe fast factorization.") << endl;
		}
		cout << endl << endl;
	}

	return true;
}


// -----------------------------------------------------------------------------
// ShowUsage()
//
// This function displays the correct usage of this program
// -----------------------------------------------------------------------------
void ShowUsage()
{
	cout << "Usage:  driver_mm [OPTIONS]" << endl;
	cout << endl;
	cout << " -b SIZE BW DD" << endl;
	cout << " --banded-synthetic SIZE BW DD" << endl;
	cout << "        Use a synthetic banded matrix of size SIZE, half-bandwidth BW," << endl;
	cout << "        and degree of diagonal dominance DD." << endl;
	cout << " -o=OUTFILE" << endl;
	cout << " --output-file=OUTFILE" << endl;
	cout << "        Write the solution to the file OUTFILE (MatrixMarket format)." << endl;
	cout << " -p=NUM_PARTITIONS" << endl;
	cout << " --num-partitions=NUM_PARTITIONS" << endl;
	cout << "        Specify the number of partitions (default 1)." << endl;
	cout << " -t=TOLERANCE" << endl;
	cout << " --tolerance=TOLERANCE" << endl;
	cout << " --relTol=TOLERANCE" << endl;
	cout << "        Use relative tolerance TOLERANCE for Krylov stopping criteria (default 1e-6)." << endl;
	cout << " --absTol=TOLERANCE" << endl;
	cout << "        Use absolute tolerance TOLERANCE for Krylov stopping criteria (default 0)." << endl;
	cout << " -i=ITERATIONS" << endl;
	cout << " --max-num-iterations=ITERATIONS" << endl;
	cout << "        Use at most ITERATIONS for Krylov solver (default 100)." << endl;
	cout << " -d=FRACTION" << endl;
	cout << " --drop-off-fraction=FRACTION" << endl;
	cout << "        Drop off-diagonal elements such that FRACTION of the matrix" << endl;
	cout << "        Frobenius norm is ignored (default 0.0 -- i.e. no drop-off)." << endl;
	cout << " --safe-fact" << endl;
	cout << "        Use safe LU-UL factorization." << endl; 
	cout << " -k=METHOD" << endl;
	cout << " --krylov-method=METHOD" << endl;
	cout << "        Specify the iterative Krylov solver:" << endl;
	cout << "        METHOD=0 or METHOD=BICGSTAB_C    use BiCGStab (Cusp)" << endl;
	cout << "        METHOD=1 or METHOD=GMRES_C       use GMRES (Cusp)" << endl;
	cout << "        METHOD=2 or METHOD=CG_C          use CG (Cusp)" << endl;
	cout << "        METHOD=3 or METHOD=CR_C          use CR (Cusp)" << endl;
	cout << "        METHOD=4 or METHOD=BICGSTAB1     use BiCGStab(1) (Spike::GPU)" << endl;
	cout << "        METHOD=5 or METHOD=BICGSTAB2     use BiCGStab(2) (Spike::GPU). This is the default." << endl;
	cout << "        METHOD=6 or METHOD=BICGSTAB      use BiCGStab (Spike::GPU)" << endl;
	cout << "        METHOD=7 or METHOD=MINRES        use MINRES (Spike::GPU)" << endl;
	cout << " --precond-method=METHOD" << endl;
	cout << "        Specify the preconditioner to be used" << endl;
	cout << "        METHOD=0 or METHOD=SPIKE         SPIKE preconditioner.  This is the default." << endl;
	cout << "        METHOD=1 or METHOD=BLOCK         Block-diagonal preconditioner." << endl;
	cout << "        METHOD=2 or METHOD=NONE          no preconditioner." << endl;
	cout << " -? -h --help" << endl;
	cout << "        Print this message and exit." << endl;
	cout << endl;
}


// -------------------------------------------------------------------
// GetBandedMatrix()
//
// This function generates a banded matrix of specified size, half
// bandwidth, and degree of diagonal dominance. The matrix is first
// generated on a local COO matrix on the host and is then copied to
// the output matrix. We use random elements in the range [-10, 10]
// and adjust the diagonal elements to satisfy the required degree of
// diagonal dominance.
// -------------------------------------------------------------------
void
GetBandedMatrix(int N, int k, REAL d, Matrix& A)
{
	// Generate the banded matrix (in COO format) on the host.
	int     num_entries = (2 * k + 1) * N - k * (k + 1);
	MatrixCooH Ah(N, N, num_entries);

	int iiz = 0;
	for (int ir = 0; ir < N; ir++) {
		int left = std::max(0, ir - k);
		int right = std::min(N - 1, ir + k);

		REAL row_sum = 0;
		int  diag_iiz;
		for (int ic = left; ic <= right; ic++, iiz++) {
			REAL val = RAND(-10.0, 10.0);////(ir+1)*(ic+1);

			if (ir == ic)
				diag_iiz = iiz;
			else
				row_sum += abs(val);

			Ah.row_indices[iiz] = ir;
			Ah.column_indices[iiz] = ic;
			Ah.values[iiz] = val;
		}
		Ah.values[diag_iiz] = d * row_sum;
	}

	// Copy the matrix from host to device, while also converting it 
	// from COO to CSR format.
	A = Ah;

	////cusp::io::write_matrix_market_file(Ah, "A.mtx");
}


// -----------------------------------------------------------------------------
// PrintStats()
//
// This function prints solver statistics.
// -----------------------------------------------------------------------------
void PrintStats(bool               success,
                const SpikeSolver& mySolver,
                const SpmvFunctor& mySpmv)
{
	spike::Stats stats = mySolver.getStats();

	cout << endl;
	cout << (success ? "Success" : "Failed") << endl;

	cout << "Code: " << mySolver.getMonitorCode();
	cout << "  " << mySolver.getMonitorMessage() << endl;

	cout << "Number of iterations = " << stats.numIterations << endl;
	cout << "RHS norm             = " << stats.rhsNorm << endl;
	cout << "Residual norm        = " << stats.residualNorm << endl;
	cout << "Rel. residual norm   = " << stats.relResidualNorm << endl;
	cout << endl;
	cout << "Bandwidth                  = " << stats.bandwidth << endl;
	cout << "Actual drop-off fraction   = " << stats.actualDropOff << endl;
	cout << endl;
	cout << "Setup time total  = " << stats.timeSetup << endl;
	double timeSetupGPU = stats.time_toBanded + stats.time_offDiags
		+ stats.time_bandLU + stats.time_bandUL
		+ stats.time_assembly + stats.time_fullLU;
	cout << "  Setup time GPU  = " << timeSetupGPU << endl;
	cout << "    form banded matrix       = " << stats.time_toBanded << endl;
	cout << "    extract off-diags blocks = " << stats.time_offDiags << endl;
	cout << "    banded LU factorization  = " << stats.time_bandLU << endl;
	cout << "    banded UL factorization  = " << stats.time_bandUL << endl;
	cout << "    assemble reduced matrix  = " << stats.time_assembly << endl;
	cout << "    reduced matrix LU        = " << stats.time_fullLU << endl;
	cout << "  Setup time CPU  = " << stats.timeSetup - timeSetupGPU << endl;
	cout << "    reorder                  = " << stats.time_reorder << endl;
	cout << "    CPU assemble             = " << stats.time_cpu_assemble << endl;
	cout << "    data transfer            = " << stats.time_transfer << endl;
	cout << "Solve time        = " << stats.timeSolve << endl;
	cout << "  shuffle time    = " << stats.time_shuffle << endl;
	cout << endl;
	cout << "SPMV count = " << mySpmv.getCount() 
		 << "  total time = " << mySpmv.getTime() 
		 << "  GFlop/s = " << mySpmv.getGFlops()
		 << endl;
	cout << endl;
}
