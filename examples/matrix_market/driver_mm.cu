#include "hip/hip_runtime.h"
#include <algorithm>

#include <cusp/io/matrix_market.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>

#include <spike/solver.h>



// Macro to obtain a random number between two specified values
#define RAND(L,H)  ((L) + ((H)-(L)) * (float)rand()/(float)RAND_MAX)
#define MAX(A,B)   (((A) > (B)) ? (A) : (B))
#define MIN(A,B)   (((A) < (B)) ? (A) : (B))


// -------------------------------------------------------------------
// Typedefs
// -------------------------------------------------------------------
typedef double REAL;

typedef cusp::device_memory MEMORY;
typedef typename cusp::csr_matrix<int, REAL, MEMORY> Matrix;
typedef typename cusp::array1d<REAL, MEMORY>         Vector;

typedef typename cusp::coo_matrix<int, REAL, cusp::host_memory>  MatrixHost;
typedef typename cusp::array1d<REAL, cusp::host_memory>          VectorHost;

typedef typename spike::Solver<Matrix, Vector>       SpikeSolver;
typedef typename spike::SpmvCusp<Matrix, Vector>     SpmvFunctor;


// -------------------------------------------------------------------
// -------------------------------------------------------------------
using std::cout;
using std::cin;
using std::endl;
using std::string;
using std::vector;


// -------------------------------------------------------------------
// Definitions for SimpleOpt and SimpleGlob
// -------------------------------------------------------------------
#include <SimpleOpt/SimpleOpt.h>


// ID values to identify command line arguments
enum {OPT_HELP, OPT_VERBOSE, OPT_PART,
	  OPT_NO_REORDERING, OPT_NO_SCALING,
	  OPT_TOL, OPT_MAXIT,
	  OPT_DROPOFF_FRAC, OPT_DROPOFF_K,
	  OPT_MATFILE, OPT_RHSFILE, 
	  OPT_OUTFILE, OPT_FACTORIZATION, OPT_PRECOND,
	  OPT_KRYLOV, OPT_SAFE_FACT,
	  OPT_VAR_BAND, OPT_SECOND_REORDER, OPT_TRACK_REORDER,
	  OPT_SINGLE_COMP};

// Table of CSimpleOpt::Soption structures. Each entry specifies:
// - the ID for the option (returned from OptionId() during processing)
// - the option as it should appear on the command line
// - type of the option
// The last entry must be SO_END_OF_OPTIONS
CSimpleOptA::SOption g_options[] = {
	{ OPT_PART,          "-p",                   SO_REQ_CMB },
	{ OPT_PART,          "--num-partitions",     SO_REQ_CMB },
	{ OPT_TOL,           "-t",                   SO_REQ_CMB },
	{ OPT_TOL,           "--tolerance",          SO_REQ_CMB },
	{ OPT_MAXIT,         "-i",                   SO_REQ_CMB },
	{ OPT_MAXIT,         "--max-num-iterations", SO_REQ_CMB },
	{ OPT_DROPOFF_FRAC,  "-d",                   SO_REQ_CMB },
	{ OPT_DROPOFF_FRAC,  "--drop-off-fraction",  SO_REQ_CMB },
	{ OPT_DROPOFF_K,     "--drop-off-k",         SO_REQ_CMB },
	{ OPT_MATFILE,       "-m",                   SO_REQ_CMB },
	{ OPT_MATFILE,       "--matrix-file",        SO_REQ_CMB },
	{ OPT_RHSFILE,       "-r",                   SO_REQ_CMB },
	{ OPT_RHSFILE,       "--rhs-file",           SO_REQ_CMB },
	{ OPT_OUTFILE,       "-o",                   SO_REQ_CMB },
	{ OPT_OUTFILE,       "--output-file",        SO_REQ_CMB },
	{ OPT_SINGLE_COMP,	 "--single-component",	 SO_NONE	},
	{ OPT_NO_REORDERING, "-x",                   SO_NONE    },
	{ OPT_NO_REORDERING, "--no-reordering",      SO_NONE    },
	{ OPT_NO_SCALING,    "-y",                   SO_NONE    },
	{ OPT_NO_SCALING,    "--no-scaling",         SO_NONE    },
	{ OPT_FACTORIZATION, "-f",                   SO_REQ_CMB },
	{ OPT_FACTORIZATION, "--factorization-method", SO_REQ_CMB },
	{ OPT_PRECOND,		 "--precond-method",	 SO_REQ_CMB },
	{ OPT_SECOND_REORDER,"--second-reorder",	 SO_NONE	},
	{ OPT_KRYLOV,        "-k",                   SO_REQ_CMB },
	{ OPT_KRYLOV,        "--krylov-method",      SO_REQ_CMB },
	{ OPT_SAFE_FACT,     "--safe-fact",          SO_NONE    },
	{ OPT_VAR_BAND,      "--var-band",           SO_NONE    },
	{ OPT_TRACK_REORDER, "--track-reorder",		 SO_NONE	},
	{ OPT_VERBOSE,       "-v",                   SO_NONE    },
	{ OPT_VERBOSE,       "--verbose",            SO_NONE    },
    { OPT_HELP,          "-?",                   SO_NONE    },
	{ OPT_HELP,          "-h",                   SO_NONE    },
    { OPT_HELP,          "--help",               SO_NONE    },
	SO_END_OF_OPTIONS
};


// -------------------------------------------------------------------
// Problem types
// Problem definition
// -------------------------------------------------------------------
struct Problem {
	int           N;
	int           k;
	REAL          d;

	int           numPart;

	int           maxIt;
	REAL          tol;
	REAL          fraction;
	int           dropped;

	bool          reorder;
	bool          scale;

	string        fileMat;
	string        fileRhs;
	string        fileSol;

	spike::SolverType    krylov;
	spike::SolverMethod  factorization;
	spike::PrecondMethod precondMethod;

	bool		  singleComponent;
	bool          safeFactorization;
	bool		  variousBandwidth;
	bool		  secondLevelReordering;
	bool		  trackReordering;

	bool          verbose;
};


// -------------------------------------------------------------------
// Forward declarations.
// -------------------------------------------------------------------
void ShowUsage();
static const char* GetLastErrorText(int a_nError) ;

void spikeSetDevice();

bool GetProblemSpecs(int argc, char** argv, Problem& pb);

void GetRhsVector(const Matrix& A, Vector& b, Vector& x_target);

void PrintProblem(const Problem& pb, bool verbose);
void PrintStats(bool               success,
                const SpikeSolver& mySolver,
                const SpmvFunctor& mySpmv,
                bool               verbose);

void ClearStats(const SpikeSolver& mySolver);


// -------------------------------------------------------------------
// MAIN
// -------------------------------------------------------------------
int main(int argc, char** argv) 
{
	Problem pb;

	pb.N = 0;
	pb.k = 0;
	pb.d = 1.0;
	pb.maxIt = 100;
	pb.tol = 1e-6;
	pb.fraction = 0.0;
	pb.dropped = 0;
	pb.reorder = true;
	pb.scale = true;
	pb.numPart = 1;

	pb.factorization = spike::LU_UL;
	pb.precondMethod = spike::Spike;
	pb.singleComponent = false;
	pb.safeFactorization = false;
	pb.variousBandwidth = false;
	pb.secondLevelReordering = false;
	pb.krylov = spike::BiCGStab2;

	pb.verbose = false;


	// Get the problem specification from the program arguments.
	if (!GetProblemSpecs(argc, argv, pb))
		return 1;

	if (!pb.reorder) {
		pb.secondLevelReordering  = false;
		pb.variousBandwidth = false;
	}

	if (pb.variousBandwidth) {
		pb.secondLevelReordering  = true;
		pb.factorization = spike::LU_only;
	} else if (pb.secondLevelReordering) {
		pb.variousBandwidth = true;
		pb.factorization = spike::LU_only;
	}

	// Print information on the problem that will be solved.
	PrintProblem(pb, pb.verbose);


	// Get the device with most available memory.
	spikeSetDevice();


	// Get matrix and rhs. Note that the 'target' solution is only
	// set if using a generated RHS.
	Matrix A;
	Vector b;
	Vector x_target;
	Vector delta_x_target;

	cusp::io::read_matrix_market_file(A, pb.fileMat);
	pb.N = A.num_rows;
	if (pb.fileRhs.length() > 0)
		cusp::io::read_matrix_market_file(b, pb.fileRhs);
	else
		GetRhsVector(A, b, x_target);


	// Create the SPIKE Solver object and the SPMV functor.
	// Set the initial guess to the zero vector.
	SpikeSolver  mySolver(pb.numPart, pb.maxIt, pb.tol, pb.reorder, pb.scale, pb.fraction, pb.dropped, pb.krylov, pb.factorization, pb.precondMethod, pb.singleComponent, pb.safeFactorization, pb.variousBandwidth, pb.secondLevelReordering, pb.trackReordering);
	for (int i=0; i<1; i++) {
		if (i > 0) {
			cusp::blas::axpy(A.values, A.values, 0.05);
			cusp::blas::axpy(b, b, 0.05);
		}

		SpmvFunctor  mySpmv(A);
		Vector       x(pb.N, 0);

		mySolver.setup(A);
		bool success = mySolver.solve(mySpmv, b, x);

		// If an output file was specified, write the solution vector
		// in MatrixMarket format.
		if (pb.fileSol.length() > 0)
			cusp::io::write_matrix_market_file(x, pb.fileSol);

		// Print solution statistics.
		PrintStats(success, mySolver, mySpmv, pb.verbose);

		// Calculate the actual residual and its norm.
		Vector r(pb.N);
		mySpmv(x, r);
		cusp::blas::axpby(b, r, r, REAL(1.0), REAL(-1.0));
		cout << "|b - A*x|      = " << cusp::blas::nrm2(r) << endl;
		cout << "|b|            = " << cusp::blas::nrm2(b) << endl;	

		// If we have used a generated RHS, print the difference
		// between the target solution and the obtained solution.
		//    x_target <- x_target - x
		if (pb.fileRhs.length() == 0) {
			cout << "|x_target|     = " << cusp::blas::nrm2(x_target) << endl;
			delta_x_target.resize(pb.N);
			cusp::blas::axpby(x_target, x, delta_x_target, REAL(1.0), REAL(-1.0));
			cout << "|x_target - x| = " << cusp::blas::nrm2(delta_x_target) << endl;
		}

		ClearStats(mySolver);
	}

	// That's all folks!
	return 0;
}

// -------------------------------------------------------------------
// spikeSetDevice()
//
// This function gets the device with maximum free space and set that
// device to working device. 
// FIXME:
// Note that this function shall be removed when we start multi-gpu
// support. 
// -------------------------------------------------------------------
void spikeSetDevice() {
	int deviceCount = 0;
	
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		hipSetDevice(0);
		return;
	}

	size_t max_free_size = 0;
	int max_idx = 0;
	for (int i=0; i < deviceCount; i++) {
		hipSetDevice(i);
		size_t free_size = 0, total_size = 0;
		if (hipMemGetInfo(&free_size, &total_size) == hipSuccess)
			if (max_free_size < free_size) {
				max_idx = i;
				max_free_size = free_size;
			}
	}

	fprintf(stderr, "Use Device: %d\n", max_idx);
	hipSetDevice(max_idx);
}

// -------------------------------------------------------------------
// GetProblemSpecs()
//
// This function parses the specified program arguments and sets up
// the problem to be solved.
// -------------------------------------------------------------------
bool
GetProblemSpecs(int argc, char** argv, Problem& pb)
{
	// Create the option parser and pass it the arguments from main
	// and the array of valid options.
	CSimpleOptA args(argc, argv, g_options);

	// Loop while there are arguments to be processed.
	while (args.Next()) {
		// Exit immediately if we encounter an invalid argument.
		if (args.LastError() != SO_SUCCESS) {
			cout << "Invalid argument: " << args.OptionText() << endl;
			ShowUsage();
			return false;
		}

		switch (args.OptionId()) {

		case OPT_HELP:
			ShowUsage();
			return false;

		case OPT_PART:
			pb.numPart = atoi(args.OptionArg());
			if (pb.numPart <= 0) {
				cout << "Invalid value for the number of partitions. P = " << pb.numPart << endl;
				return false;
			}
			break;

		case OPT_TOL:
			pb.tol = atof(args.OptionArg());
			break;
			
		case OPT_MAXIT:
			pb.maxIt = atoi(args.OptionArg());
			break;

		case OPT_DROPOFF_FRAC:
			pb.fraction = atof(args.OptionArg());
			break;

		case OPT_DROPOFF_K:
			pb.dropped = atoi(args.OptionArg());
			break;

		case OPT_NO_REORDERING:
			pb.reorder = false;
			break;

		case OPT_SECOND_REORDER:
			pb.secondLevelReordering = true;
			break;

		case OPT_NO_SCALING:
			pb.scale = false;
			break;

		case OPT_VERBOSE:
			pb.verbose = true;
			break;

		case OPT_MATFILE:
			pb.fileMat = args.OptionArg();

			break;

		case OPT_RHSFILE:
			pb.fileRhs = args.OptionArg();
			break;

		case OPT_OUTFILE:
			pb.fileSol = args.OptionArg();
			break;

		case OPT_FACTORIZATION:
			{
				string fact = args.OptionArg();
				std::transform(fact.begin(), fact.end(), fact.begin(), ::toupper);
				if (fact == "0" || fact == "LU_UL")
					pb.factorization = spike::LU_UL;
				else if (fact == "1" || fact == "LU_LU")
					pb.factorization = spike::LU_only;
				else
					return false;
			}

			break;

		case OPT_PRECOND:
			{
				string precond = args.OptionArg();
				std::transform(precond.begin(), precond.end(), precond.begin(), ::toupper);
				if (precond == "0" || precond == "SPIKE")
					pb.precondMethod = spike::Spike;
				else if(precond == "1" || precond == "BLOCK")
					pb.precondMethod = spike::Block;
				else
					return false;
			}

			break;

		case OPT_KRYLOV:
			{
				string kry = args.OptionArg();
				std::transform(kry.begin(), kry.end(), kry.begin(), ::toupper);
				if (kry == "0" || kry == "BICGSTAB")
					pb.krylov = spike::BiCGStab;
				else if (kry == "1" || kry == "BICGSTAB2")
					pb.krylov = spike::BiCGStab2;
				else
					return false;
			}

			break;

		case OPT_SINGLE_COMP:
			pb.singleComponent = true;
			break;

		case OPT_SAFE_FACT:
			pb.safeFactorization = true;
			break;

		case OPT_VAR_BAND:
			pb.variousBandwidth = true;
			break;
		case OPT_TRACK_REORDER:
			pb.trackReordering = true;
			break;
		}

	}

	// If no problem was defined, show usage and exit.
	if (pb.fileRhs.length() == 0) {
		cout << "No matrix file was defined!" << endl << endl;
		ShowUsage();
		return false;
	}

	return true;
}


// -------------------------------------------------------------------
// ShowUsage()
//
// This function displays the correct usage of this program
// -------------------------------------------------------------------
void ShowUsage()
{
	cout << "Usage:  tSpike [OPTIONS]" << endl;
	cout << endl;
	cout << " -p=NUM_PARTITIONS" << endl;
	cout << " --num-partitions=NUM_PARTITIONS" << endl;
	cout << "        Specify the number of partitions (default 1)." << endl;
	cout << " -x" << endl;
	cout << " --no-reordering" << endl;
	cout << "        Do not perform reordering." << endl;
	cout << " -y" << endl;
	cout << " --no-scaling" << endl;
	cout << "        Do not perform scaling (ignored if --no-reordering is specified)" << endl;
	cout << " -t=TOLERANCE" << endl;
	cout << " --tolerance=TOLERANCE" << endl;
	cout << "        Use TOLERANCE for BiCGStab stopping criteria (default 1e-6)." << endl;
	cout << " -i=ITERATIONS" << endl;
	cout << " --max-num-iterations=ITERATIONS" << endl;
	cout << "        Use at most ITERATIONS for BiCGStab (default 100)." << endl;
	cout << " -d=FRACTION" << endl;
	cout << " --drop-off-fraction=FRACTION" << endl;
	cout << "        Drop off-diagonal elements such that FRACTION of the matrix" << endl;
	cout << "        Frobenius norm is ignored (default 0.0 -- i.e. no drop-off)." << endl;
	cout << " --drop-off-k=K" << endl;
	cout << "        Drop K pairs of off-diagonals. Ignored if --drop-off-fraction" << endl;
	cout << "        is specified. (default 0 -- i.e. no drop-off)." << endl;
	cout << " -m=MATFILE" << endl;
	cout << " --matrix-file=MATFILE" << endl;
	cout << "        Read the matrix from the file MATFILE (MatrixMarket format)." << endl;
	cout << " -r=RHSFILE" << endl;
	cout << " --rhs-file=RHSFILE" << endl;
	cout << "        Read the right-handside vector from the file RHSFILE (MatrixMarket format)." << endl;
	cout << "        Only used if '-m' is specified." << endl;
	cout << " -o=OUTFILE" << endl;
	cout << " --output-file=OUTFILE" << endl;
	cout << "        Write the solution to the file OUTFILE (MatrixMarket format)." << endl;
	cout << " -k=METHOD" << endl;
	cout << " --krylov-method=METHOD" << endl;
	cout << "        Specify the iterative Krylov solver:" << endl;
	cout << "        METHOD=0 or METHOD=bicgstab      use BiCGStab" << endl;
	cout << "        METHOD=1 or METHOD=bicgstab2     use BiCGStab(2). This is the default." << endl;
	cout << " --safe-fact" << endl;
	cout << "        Use safe LU-UL factorization." << endl; 
	cout << " --var-band" << endl;
	cout << "        Use various-bandwidth-method to solve the problem." << endl; 
	cout << " --second-reorder" << endl;
	cout << "        Use second-level reordering." << endl;
	cout << " -f=METHOD" << endl;
	cout << " --factorization-method=METHOD" << endl;
	cout << "        Specify the factorization type used to assemble the reduced matrix" << endl;
	cout << "        METHOD=0 or METHOD=lu_ul                for both applying LU and UL.  This is the default." << endl;
	cout << "        METHOD=1 or METHOD=lu_lu                for applying a complete LU" << endl;
	cout << " --precond-method=METHOD" << endl;
	cout << "        Specify the preconditioner to be used" << endl;
	cout << "        METHOD=0 or METHOD=SPIKE                for using SPIKE preconditioner.  This is the default." << endl;
	cout << "        METHOD=1 or METHOD=BLOCK                for using Block preconditionera." << endl;
	cout << " -v --verbose" << endl;
	cout << "        Verbose output." << endl; 
	cout << " -? -h --help" << endl;
	cout << "        Print this message and exit." << endl;
	cout << endl;
}


// -------------------------------------------------------------------
// GetLastErrorText()
//
// This function translates SO error codes to human readable strings.
// -------------------------------------------------------------------
static const char* GetLastErrorText(int a_nError) 
{
    switch (a_nError) {
    case SO_SUCCESS:            return "Success";
    case SO_OPT_INVALID:        return "Unrecognized option";
    case SO_OPT_MULTIPLE:       return "Option matched multiple strings";
    case SO_ARG_INVALID:        return "Option does not accept argument";
    case SO_ARG_INVALID_TYPE:   return "Invalid argument format";
    case SO_ARG_MISSING:        return "Required argument is missing";
    case SO_ARG_INVALID_DATA:   return "Invalid argument data";
    default:                    return "Unknown error";
    }
}


// -------------------------------------------------------------------
// GetRhsVector()
//
// This function generates a RHS vector of appropriate dimension. We
// use the method of manufactured solution, meaning we set
//    b = A * x
// for a known "solution" vector x.
// -------------------------------------------------------------------
void
GetRhsVector(const Matrix& A, Vector& b, Vector& x_target)
{
	// Create a desired solution vector (on the host), then copy it
	// to the device.
	int     N = A.num_rows;
	REAL    dt = 1.0/(N-1);
	REAL    max_val = 100.0;

	VectorHost xh(N);

	for (int i = 0; i < N; i++) {
		REAL t = i *dt;
		xh[i] = 4 * max_val * t * (1 - t);
	}

	x_target = xh;
	
	// Calculate the RHS vector.
	b.resize(N);
	cusp::multiply(A, x_target, b);
	////cusp::io::write_matrix_market_file(b, "b.mtx");
}


// -------------------------------------------------------------------
// PrintProblem()
//
// This function prints information about the problem and solution
// parameters.
// -------------------------------------------------------------------
void PrintProblem(const Problem& pb, bool verbose)
{
	string factTypeStr;
	string precondStr;
	switch (pb.factorization) {
	case spike::LU_UL:
		factTypeStr = "LU-UL";
		break;
	case spike::LU_only:
		factTypeStr = "LU-LU";
		break;
	}

	switch (pb.precondMethod) {
	case spike::Spike:
		precondStr = "SPIKE";
		break;
	case spike::Block:
		precondStr = "BLOCK";
		break;
	}

	string krylovTypeStr;
	switch (pb.krylov) {
	case spike::BiCGStab:
		krylovTypeStr = "BiCGStab";
		break;
	case spike::BiCGStab2:
		krylovTypeStr = "BiCGStab2";
		break;
	}

	if (verbose) {
		cout << endl;
		cout << "Application problem from file" << endl
			<< "  Matrix file: " << pb.fileMat << endl;
		if (pb.fileRhs.length() > 0)
			cout << "  Rhs file: " << pb.fileRhs << endl;
		cout << "Using " << pb.numPart << " partition";
		if (pb.numPart > 1)
			cout << "s";
		cout << "." << endl;
		cout << "Iterative solver: " << krylovTypeStr << endl;
		cout << "Tolerance: " << pb.tol << endl;
		cout << "Max. iterations: " << pb.maxIt << endl;
		if (pb.fraction > 0)
			cout << "Drop-off fraction: " << pb.fraction << endl;
		else if (pb.dropped > 0)
			cout << "Drop off-diagonals: " << pb.dropped << endl;
		else
			cout << "No drop-off." << endl;
		cout << (pb.singleComponent ? "Do not break the problem into several components." : "Attempt to break the problem into several components.") << endl;
		cout << (pb.reorder ? "Perform reordering." : "Do not perform reordering.") << endl;
		cout << (pb.scale   ? "Apply scaling." : "Do not apply scaling.") << endl;
		cout << (pb.safeFactorization ? "Using safe factorization." : "Using non-safe fast factorization.") << endl;
		cout << (pb.variousBandwidth ? "Using various-bandwidth method." : "Not using various-bandwidth method.") << endl;
		cout << "Factorization method: " << factTypeStr << endl;
		cout << "Preconditioner: " << precondStr << endl;
		if (pb.fileRhs.length() > 0)
			cout << "Sol file: " << pb.fileSol << endl;
		cout << endl << endl;
	} else {
		cout << pb.fileMat;
		cout << "     " << pb.numPart;
		cout << "     " << pb.maxIt << "  " << pb.tol;
		cout << "     " << (pb.reorder ? "T" : "F") << (pb.scale ? "T" : "F");
		cout << "     " << pb.fraction;
		cout << "     " << factTypeStr << "  " << precondStr << "   "<<(pb.safeFactorization ? "T" : "F");
		cout << "	  " << (pb.variousBandwidth ? "T" : "F");
		cout << "	  " << (pb.secondLevelReordering? "T" : "F");
		cout << endl;
	}
}


// -------------------------------------------------------------------
// PrintStats()
//
// This function prints solver statistics.
// -------------------------------------------------------------------
void PrintStats(bool               success,
                const SpikeSolver& mySolver,
                const SpmvFunctor& mySpmv,
                bool               verbose)
{
	spike::SolverStats stats = mySolver.getStats();

	if (verbose) {
		cout << endl;
		cout << (success ? "Success" : "Failed") << endl;

		cout << "Number of iterations = " << stats.numIterations << endl;
		cout << "Residual norm        = " << stats.residualNorm << endl;
		cout << "Rel. residual norm   = " << stats.relResidualNorm << endl;
		cout << endl;
		cout << "Bandwidth after reordering = " << stats.bandwidthReorder << endl;
		cout << "Bandwidth                  = " << stats.bandwidth << endl;
		cout << "Actual drop-off fraction   = " << stats.actualDropOff << endl;
		cout << endl;
		cout << "Setup time total  = " << stats.timeSetup << endl;
		double timeSetupGPU = stats.time_toBanded + stats.time_offDiags
			+ stats.time_bandLU + stats.time_bandUL
			+ stats.time_assembly + stats.time_fullLU;
		cout << "Setup time GPU    = " << timeSetupGPU << endl;
		cout << "  form banded matrix       = " << stats.time_toBanded << endl;
		cout << "  extract off-diags blocks = " << stats.time_offDiags << endl;
		cout << "  banded LU factorization  = " << stats.time_bandLU << endl;
		cout << "  banded UL factorization  = " << stats.time_bandUL << endl;
		cout << "  assemble reduced matrix  = " << stats.time_assembly << endl;
		cout << "  reduced matrix LU        = " << stats.time_fullLU << endl;
		cout << "Solve time        = " << stats.timeSolve << endl;
		cout << "  shuffle time    = " << stats.time_shuffle << endl;
		cout << endl;
		cout << "SPMV count = " << mySpmv.getCount() 
			 << "  total time = " << mySpmv.getTime() 
			 << "  GFlop/s = " << mySpmv.getGFlops()
			 << endl;
		cout << endl;
	} else {
		cout << (success ? "T  " : "F  ");
		cout << stats.numIterations    << "  ";
		cout << stats.residualNorm     << "  ";
		cout << stats.relResidualNorm  << "    ";
		cout << stats.timeSetup        << "  ";
		cout << stats.timeSolve        << "  ";
		cout << stats.time_bandLU      << "  ";
		cout << stats.time_bandUL      << "  ";
		cout << stats.time_fullLU      << "    ";
		cout << mySpmv.getCount()      << "  ";
		cout << mySpmv.getTime()       << "  ";
		cout << endl;
	}
}

// -------------------------------------------------------------------
// ClearStats()
//
// This function clears solver statistics.
// -------------------------------------------------------------------
void ClearStats(const SpikeSolver& mySolver)
{
	spike::SolverStats stats = mySolver.getStats();

	stats.numIterations = 0;
	stats.residualNorm = 0;
	stats.bandwidthReorder = 0;
	stats.bandwidth = 0;
	stats.actualDropOff = 0;
	stats.timeSetup = 0;
	stats.time_toBanded = 0;
	stats.time_offDiags = 0;
	stats.time_bandLU = 0;
	stats.time_bandUL = 0;
	stats.time_assembly = 0;
	stats.time_fullLU = 0;
	stats.timeSolve = 0;
	stats.time_shuffle = 0;
}

